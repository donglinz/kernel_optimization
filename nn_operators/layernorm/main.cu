//
// Created by dongl on 5/13/2022.
//

#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime.h>
#include "layernorm.h"
#include "../../common/tensor.h"
#include "../../common/common.h"
#include <random>

template<
        typename T,
        template<typename, int, int, int, bool> typename LayerNorm,
        int vec_len,
        int num_row,
        int num_column>
void run_benchmark() {
    using LayerNormKernel = LayerNorm<T, vec_len, num_row, num_column, false>;

    Tensor in_data(num_row * num_column * sizeof(T));
    Tensor out_data(num_row * num_column * sizeof(T));

    hipStream_t stream;
    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    std::default_random_engine generator;
    std::normal_distribution<double> dist(0, 1);

    for (int idx = 0; idx < num_row * num_column; ++idx) {
        in_data.template host_ref<T>()[idx] = T(dist(generator));
    }

    in_data.host_to_device_async(stream);

    if (LayerNormKernel::smem_in_bytes >= (48 << 10)) {
        checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(layer_normalization<LayerNormKernel>), hipFuncAttributeMaxDynamicSharedMemorySize, LayerNormKernel::smem_in_bytes));
    }

    // warp up
    for (int idx = 0; idx < 10; ++idx) {
        layer_normalization<LayerNormKernel>
        <<<108*4, 128, LayerNormKernel::smem_in_bytes, stream>>>
        (in_data.template device_ref<T>(), out_data.template device_ref<T>(), 0.0001);
    }

    checkCudaErrors(hipStreamSynchronize(stream));

    hipEvent_t _start, _stop;
    checkCudaErrors(hipEventCreate(&_start));
    checkCudaErrors(hipEventCreate(&_stop));

    float time_in_ms = 0;

    for (int idx = 0; idx < 10; ++idx) {
        in_data.host_to_device_async(stream);

        checkCudaErrors(hipEventRecord(_start, stream));
        layer_normalization<LayerNormKernel>
        <<<108*4, 128, LayerNormKernel::smem_in_bytes, stream>>>
        (in_data.template device_ref<T>(), out_data.template device_ref<T>(), 0.0001);
        checkCudaErrors(hipEventRecord(_stop, stream));

        checkCudaErrors(hipEventSynchronize(_stop));

        float ms;
        checkCudaErrors(hipEventElapsedTime(&ms, _start, _stop));
        time_in_ms += ms;
    }

    checkCudaErrors(hipStreamSynchronize(stream));

    printf("======Num row %d, num column %d======\n", num_row, num_column);
    printf("Average time %.2fms\n", time_in_ms / 10);
}
int main() {
    run_benchmark<float, LayerNormWarpImpl, 4, 49152, 1024>();
    run_benchmark<float, LayerNormBlockImpl, 4, 49152, 1024>();
    run_benchmark<float, LayerNormBlockNoCacheImpl, 4, 49152, 1024>();
    return 0;
}