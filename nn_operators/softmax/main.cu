//
// Created by dongl on 5/13/2022.
//

#include <random>

#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime.h>
#include "softmax.h"
#include "../../common/tensor.h"
#include "../../common/common.h"

template<
        int grid_dim,
        int block_dim,
        typename T,
        template<typename, int, int, int> typename SoftMax,
        int vec_len,
        int num_row,
        int num_column>
void run_benchmark() {
    using SoftMaxKernel = SoftMax<T, vec_len, num_row, num_column>;

    Tensor in_data(num_row * num_column * sizeof(T));
    Tensor out_data(num_row * num_column * sizeof(T));

    hipStream_t stream;
    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    std::default_random_engine generator;
    std::normal_distribution<double> dist(0, 1);

    for (int idx = 0; idx < num_row * num_column; ++idx) {
        in_data.template host_ref<T>()[idx] = T(dist(generator));
    }

    in_data.host_to_device_async(stream);

    if (SoftMaxKernel::smem_in_bytes >= (48 << 10)) {
        checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(softmax<SoftMaxKernel>), hipFuncAttributeMaxDynamicSharedMemorySize, SoftMaxKernel::smem_in_bytes));
    }

    int n_iter = 1;

    checkCudaErrors(hipStreamSynchronize(stream));

    hipEvent_t _start, _stop;
    checkCudaErrors(hipEventCreate(&_start));
    checkCudaErrors(hipEventCreate(&_stop));

    float time_in_ms = 0;

    for (int idx = 0; idx < n_iter; ++idx) {
        in_data.host_to_device_async(stream);

        checkCudaErrors(hipEventRecord(_start, stream));
        softmax<SoftMaxKernel>
        <<<grid_dim, block_dim, SoftMaxKernel::smem_in_bytes, stream>>>
                (in_data.template device_ref<T>(), out_data.template device_ref<T>());
        checkCudaErrors(hipEventRecord(_stop, stream));

        checkCudaErrors(hipEventSynchronize(_stop));

        float ms;
        checkCudaErrors(hipEventElapsedTime(&ms, _start, _stop));
        time_in_ms += ms;
    }

    checkCudaErrors(hipStreamSynchronize(stream));

    printf("======Num row %d, num column %d======\n", num_row, num_column);
    printf("Average time %.2fms\n", time_in_ms / n_iter);
}

int main() {
    run_benchmark<40*8, 128, float, SoftMaxWarpImpl, 4, 128 * 12 * 128, 128>();
//    run_benchmark<float, SoftMaxBlockImpl, 4, 49152, 1024>();
//    run_benchmark<float, SoftMaxBlockNoCacheImpl, 4, 49152, 1024>();
    return 0;
}